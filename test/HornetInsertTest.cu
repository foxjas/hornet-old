#include "Hornet.hpp"
#include "Core/GPUHornet/BatchUpdate.cuh"
#include "Util/BatchFunctions.hpp"
#include <Host/FileUtil.hpp>            //xlib::extract_filepath_noextension
#include <Device/Util/CudaUtil.cuh>     //xlib::deviceInfo
#include <algorithm>                    //std:.generate
#include <chrono>                       //std::chrono
#include <random>                       //std::mt19937_64
#include <hip/hip_runtime_api.h>

using namespace hornets_nest;
using namespace timer;
using namespace std::string_literals;
using namespace gpu::batch_property;

using HornetGPU = hornets_nest::gpu::Hornet<EMPTY, EMPTY>;

void exec(int argc, char* argv[]);

/**
 * @brief Example tester for Hornet
 */
int main(int argc, char* argv[]) {
    exec(argc, argv);
    hipDeviceReset();
}

void exec(int argc, char* argv[]) {
    using namespace graph::structure_prop;
    using namespace graph::parsing_prop;
    xlib::device_info();

    graph::GraphStd<vid_t, eoff_t> graph;
    graph.read(argv[1]);
    //--------------------------------------------------------------------------
    HornetInit hornet_init(graph.nV(), graph.nE(), graph.csr_out_offsets(),
                           graph.csr_out_edges());

    HornetGPU hornet_gpu(hornet_init);
    std::cout << "------------------------------------------------" <<std::endl;
    using namespace batch_gen_property;

    vid_t* batch_src, *batch_dst;
    int batch_size = std::stoi(argv[2]);

    cuMallocHost(batch_src, batch_size);
    cuMallocHost(batch_dst, batch_size);

    generateBatch(graph,
            batch_size, batch_src, batch_dst,
            BatchGenType::INSERT);

    gpu::BatchUpdate batch_update(batch_src, batch_dst, batch_size);

    hornet_gpu.reserveBatchOpResource(batch_size);

    printf("ne: %d\n", hornet_gpu.nE());
    std::cout<<"=======\n";
    Timer<DEVICE> TM(3);
    TM.start();
    hornet_gpu.insertEdgeBatch(batch_update);

    TM.stop();

    printf("ne: %d\n", hornet_gpu.nE());
    std::cout<<"=======\n";
    TM.print("Insertion " + std::to_string(batch_size) + ":  ");

    cuFreeHost(batch_src);
    cuFreeHost(batch_dst);
}
